
#include <hip/hip_runtime.h>
//----------------------------------------------------------------------------//
// http://docs.nvidia.com/cuda/inline-ptx-assembly/index.html#axzz4evAhdxJg
// https://devtalk.nvidia.com/default/topic/527205/problem-about-inline-ptx-code-in-cuda-program/
//----------------------------------------------------------------------------//

__global__ void kernel_load_global(float *my_array, uint *start_t, uint *end_t, 
		int arraylen, float a, float b)
{
	unsigned int start_time1;                                                   
	unsigned int start_time2;                                                   
	unsigned int start_time3;                                                   

	unsigned int end_time1;                                                     
	unsigned int end_time2;                                                     
	unsigned int end_time3;                                                     

	float k;
	float *ptr_global = &my_array[0]; // ptr_global

	__syncthreads();                                                            

	start_time1 = clock();                                                      
	end_time1 = clock();                                                        

	__syncthreads();                                                            

	start_time2 = clock();                                                      
	asm volatile (                                                              
			"ld.global.f32  %0, [%1];\n\t" : "=f"(k) : "l"(ptr_global)
			);                                                                  
	end_time2 = clock();                                                        

	__syncthreads();                                                            

	start_time3 = clock();                                                      
	end_time3 = clock();                                                        

	start_t[0] = start_time1;                                                   
	start_t[1] = start_time2;                                                   
	start_t[2] = start_time3;                                                   

	end_t[0] = end_time1;                                                       
	end_t[1] = end_time2;                                                       
	end_t[2] = end_time3;                                                       

	my_array[0] = (float)k;
}


__global__ void kernel_load_shared(float *my_array, uint *start_t, uint *end_t, 
		int arraylen, float a, float b)
{
	unsigned int start_time1;
	unsigned int start_time2;
	unsigned int start_time3;

	unsigned int end_time1;
	unsigned int end_time2;
	unsigned int end_time3;

	__shared__ float sm[1];

	float k;

	// load global to shared memory
	sm[0] = my_array[0];
	__syncthreads();

	float *ptr_sm = &sm[0]; // ptr_global

	start_time1 = clock();                                                      
	end_time1 = clock();                                                        

	__syncthreads();                                                            

	start_time2 = clock();                                                      
	asm volatile (                                                              
			"ld.f32 %0, [%1];\n\t" : "=f"(k) : "l"(ptr_sm)
			);                                                                  
	end_time2 = clock();                                                        

	__syncthreads();                                                            

	start_time3 = clock();                                                      
	end_time3 = clock();                                                        

	start_t[0] = start_time1;                                                   
	start_t[1] = start_time2;                                                   
	start_t[2] = start_time3;                                                   

	end_t[0] = end_time1;                                                       
	end_t[1] = end_time2;                                                       
	end_t[2] = end_time3;                                                       

	my_array[0] = (float)k;
}
